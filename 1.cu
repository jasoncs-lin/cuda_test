#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

//----------------------------------------------
//向量加法的運算核心 (GPU) **函式前加 __global__ 即為核心, 核心只傳回 void**
__global__ void gpu_add(float* c, float* a, float* b, int n){
        for(int k=0; k<n; k++){
                c[k]=a[k]+b[k];
        }
}

//----------------------------------------------
//向量加法的一般函式 (Host)
void host_add(float* c, float* a, float* b, int n){
        for(int k=0; k<n; k++){
                c[k]=a[k]+b[k];
        }
}


//----------------------------------------------
//計算誤差用的函式
double diff(float* a, float* b, int n){
        double s=0, r=0;
        for(int k=0; k<n; k++){
                double w=a[k]-b[k];
                s+=w*w;
                r+=a[k]*a[k];
        }
        return sqrt(s/r); //相對誤差
}

//----------------------------------------------
//時間函數 (傳回單位:千分之一秒)
double ms_time(){
        return (double)clock()/CLOCKS_PER_SEC*1000.0;
}

//----------------------------------------------
//主程式
int main(){
        //設定向量大小
        int n=1024*1024;
        int size=n*sizeof(float);

        //網格與區塊設定
        int grid=1;     //gridDim  (每個網格具有的區塊數)
        int block=1;    //blockDim (每個區塊具有的執行緒數)

        //設定呼叫次數 (測量平均效能)
        int loop=100;

        //配置主機記憶體
        float *a,*b,*c,*d;
        a=(float*)malloc(size);
        b=(float*)malloc(size);
        c=(float*)malloc(size);
        d=(float*)malloc(size);

        //設定亂數的輸入向量
        srand(time(0));
        for(int k=0; k<n; k++){
                a[k]=(float)rand()/RAND_MAX*2-1;
                b[k]=(float)rand()/RAND_MAX*2-1;
        }

        //配置顯示卡記憶體
        float  *ga,*gb,*gc;
        hipMalloc((void**)&ga, size);
        hipMalloc((void**)&gb, size);
        hipMalloc((void**)&gc, size);

        //載入向量 a,b 到顯示卡記憶體中
        hipMemcpy(ga, a, size, hipMemcpyHostToDevice);
        hipMemcpy(gb, b, size, hipMemcpyHostToDevice);

        //---- part 1 : 測量精確度 --------

        //呼叫核心來運算 (GPU)
        gpu_add<<<grid, block>>>(gc, ga, gb, n);

        //呼叫一般函數來運算 (Host)
        host_add(d, a, b, n);

        //把計算結果存回主機
        hipMemcpy(c, gc, size, hipMemcpyDeviceToHost);

        //比較兩者差異
        printf("vector add N(%d) elements, diff = %g\n", n, diff(c,d,n));



        //---- part 2 : 測量效能 --------

        //測量 GPU 核心效能
        double gpu_dt = ms_time();
        for(int w=0; w<loop; w++){
                gpu_add<<<grid, block>>>(gc, ga, gb, n);
                hipDeviceSynchronize();  //避免核心執行不完全
        }
        gpu_dt = (ms_time()-gpu_dt)/loop; //平均時間


        //測量 Host 函數效能
        double host_dt = ms_time();
        for(int w=0; w<loop; w++){
                host_add(d, a, b, n);
        }
        host_dt = (ms_time()-host_dt)/loop; //平均時間


        //輸出平均執行時間
        printf("host time: %g ms\n", host_dt);
        printf("gpu  time: %g ms\n", gpu_dt);


        //釋放主機記憶體
        free(a);
        free(b);
        free(c);
        free(d);

        //釋放顯示卡記憶體
        hipFree(ga);
        hipFree(gb);
        hipFree(gc);

        return 0;
}
