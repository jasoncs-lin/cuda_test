#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
//--------------------------------------------------------------------
//A : m x l
//B : l x n
//C : m x n    (C=A*B)
//--------------------------------------------------------------------
 
void host_mm(float* C, float* A, float* B, int m, int n, int l){
 
    for(int i=0; i<m; i++)
        for(int j=0; j<n; j++) 
    {
            float s=0;
            for (int k=0; k<l; k++)
        {
                float a = A[i*l + k];
                float b = B[k*n + j];
                s += a * b;
            }
            C[i*n + j] = s;
        }
}
 
//--------------------------------------------------------------------
__global__ void gpu_mm(float* C, float* A, float* B, int m, int n, int l){
    
    //// 2D Thread ID    
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // Pvalue is used to store the element of the matrix
    // that is computed by the thread    
    float Pvalue = 0;
    
    for (int k = 0; k < l; ++k)
    { 
         float Aelement = A[ty * l + k];
         float Belement = B[k * n + tx];
         Pvalue += Aelement * Belement;
    }
    // Write the matrix to device memory;
    // each thread writes one element
    C[ty * n + tx] = Pvalue;
    //printf("threadIdx.x=%d\n", threadIdx.x);
}
 
 
//----------------------------------------------
double diff(float* a, float* b, int n){
    double s=0, r=0;
    for(int k=0; k<n; k++)
    {
        double w=a[k]-b[k];
        s+=w*w;
        r+=a[k]*a[k];
    }
    return sqrt(s/r); 
}
 
 
void random(float* a, int n){
    for(int k=0; k<n; k++){
        a[k]=(float)rand()/RAND_MAX*2-1;
    }
}
 
//----------------------------------------------
void testMatrix(int m, int n, int l)
{
    //initialize
    float *a = (float*)malloc(sizeof(float)*m*l);
    float *b = (float*)malloc(sizeof(float)*l*n);
    float *c1 = (float*)malloc(sizeof(float)*m*n);
    float *c2 = (float*)malloc(sizeof(float)*m*n);
 
    srand(time(0));
    random(a,m*l);
    random(b,l*n);
    memset(c1, 0, sizeof(float)*m*n);
    memset(c2, 0, sizeof(float)*m*n);
        
    float  *ga,*gb,*gc;
    hipMalloc((void**)&ga, m*l*sizeof(float));
    hipMalloc((void**)&gb, l*n*sizeof(float));
    hipMalloc((void**)&gc, m*n*sizeof(float));
 
    hipMemcpy(ga, a, m*l*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gb, b, l*n*sizeof(float), hipMemcpyHostToDevice);
    hipMemset(gc, 0, m*n*sizeof(float));
        
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0); 
 
    //SBMT(Single Block, Multiple Threads)
    gpu_mm<<<dim3(1,1,1), dim3(m, n, 1)>>> (gc,ga,gb,m,n,l);
    hipDeviceSynchronize();
 
    hipEventRecord(stop,0); 
    hipEventSynchronize(stop);
 
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    hipMemcpy(c2, gc, m*n*sizeof(float), hipMemcpyDeviceToHost);        
    
    double c_start,c_stop;
    double CPU_execution_time;
    c_start = (double)clock();
 
    host_mm(c1, a, b, m, n, l);
     
    c_stop = (double)clock();
    CPU_execution_time = (c_stop - c_start)/(double)CLOCKS_PER_SEC;
 
    //check precision        
    double err=diff(c1,c2,m*n);
    printf("err = %g\n", err);       
 
    printf(" ======== (Execution Infomation) ========\n");
    printf(" Excuetion Time on GPU: %3.20f s\n",elapsedTime/1000);
    printf(" Excuetion Time on CPU: %3.20f s\n",CPU_execution_time);
    printf(" Speed up = %f\n",(CPU_execution_time/(elapsedTime/1000)));
    printf(" ========================================\n\n");
 
 
    free(a); 
    free(b); 
    free(c1); 
    free(c2); 
       
    hipFree(ga);
    hipFree(gb);
    hipFree(gc);
        
}
 
//----------------------------------------------
int main()
{
 
    int m=32;
    int n=32;
    int l=32;
 
    testMatrix(m,n,l);
 
    return 0;
}
